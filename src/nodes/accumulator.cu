#include "hip/hip_runtime.h"
#include "nodes/accumulator.h"

#include "core/common_cu.h"

__global__
void AccumulatorKernel(int n, const float * __restrict__ x, float * __restrict__ out1)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i < n)
		out1[i] += x[i];
}

Accumulator::Accumulator(const deepflow::NodeParam &_block_param) : Node(_block_param) {
	LOG_IF(FATAL, _block_param.has_accumulator_param() == false) << "param.has_accumulator_param() == false";
}

void Accumulator::initForward() {	
	const deepflow::AccumulatorParam &accParam = _param.accumulator_param();
	_reset_time = (ResetTime)accParam.reset_time();
	_outputs[0]->initValue(_inputs[0]->value()->dims());
	_outputs[1]->initValue({ 1,1,1,1 });
	LOG(INFO) << "Initializing Accumulator " << _name << " - " << _outputs[0]->value()->shape();
}

void Accumulator::initBackward() {		

}

void Accumulator::forward() {
	if (_reset_time == EndOfEpoch && _context->current_iteration_per_epoch == 1) {
		DF_CUDA_CHECK(hipMemset(_outputs[0]->value()->mutableData(), 0, _outputs[0]->value()->sizeInBytes()));
		_total = 0;		
	}
	auto size = _inputs[0]->value()->size();
	AccumulatorKernel << < numOfBlocks(size), maxThreadsPerBlock >> >(size, (float*)_inputs[0]->value()->data(), (float*)_outputs[0]->value()->mutableData());
	DF_KERNEL_CHECK();
	_total += _inputs[0]->dims()[0];	
	DF_CUDA_CHECK(hipMemcpy(_outputs[1]->value()->mutableData(), &_total, sizeof(float), hipMemcpyHostToDevice));
}

void Accumulator::backward() {

}

std::string Accumulator::to_cpp() const
{
	std::string cpp = "auto " + _name + " = df.accumulator(" + _input_name_for_cpp(0) + ", ";
	if (_reset_time == EndOfEpoch)
		cpp += "Accumulator::EndOfEpoch, ";
	else if (_reset_time == Never) 
		cpp += "Accumulator::Never, ";
	cpp += "\"" + _name + "\", ";
	cpp += "{" + _to_cpp_phases() + "});";
	return cpp;
}
