#include "hip/hip_runtime.h"
#include "solvers/sgd_solver.h"
#include "core/common_cu.h"

#include <algorithm>

#include "nodes/variable.h"

__global__
void ApplyGradientKernel(const int n, const float momentum, const float learning_rate, float *w, const float *grad)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i < n) 
		w[i] = momentum * w[i] - learning_rate * grad[i];
}

SGDSolver::SGDSolver(deepflow::SolverParam *param) : Solver(param) {
	LOG_IF(FATAL, param->has_sgd_solver() == false) << "param.has_sgd_solver() == false";
	_my_param = _param->mutable_sgd_solver();
}

void SGDSolver::apply(std::shared_ptr<Variable> var) {
	auto context = var->executionContext();
	bool verbos = (context && context->debug_level > 3) ? true : false;
	if (_initialized == false) {
		LOG_IF(INFO, verbos) << "SOLVER " << name() << " FOR VARIABLE " << var->name();
		init(var);
	}	
	if (_enable_input) {
		bool is_enable = _enable_input->value()->toFloat() >= 1;
		if (!is_enable) {
			LOG_IF(INFO, verbos) << "SOLVER " << name() << " **NOT** APPLIED ON " << var->name();
			return;
		}
	}
	LOG_IF(INFO, verbos) << "APPLYING SOLVER " << name() << " ON " << var->name();
	auto size = var->output(0)->value()->size();
	ApplyGradientKernel << <numOfBlocks(size), maxThreadsPerBlock>> > (size, _my_param->momentum(), _my_param->learning_rate(), (float*)var->output(0)->value()->mutableData(), (float*)var->gradients());
	DF_KERNEL_CHECK();
	var->reset_gradients();
}

void SGDSolver::init(std::shared_ptr<Variable> var) {
	_initialized = true;
}

std::string SGDSolver::to_cpp() const
{
	std::string cpp = "auto " + name() + " = df.sgd_solver(";
	cpp += std::to_string(_my_param->momentum()) + ", ";
	cpp += std::to_string(_my_param->learning_rate()) + ", ";	
	cpp += "\"" + name() + "\"";
	cpp += ");";
	return cpp;
}
