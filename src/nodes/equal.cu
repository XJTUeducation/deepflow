#include "hip/hip_runtime.h"
#include "nodes/equal.h"

__global__
void EqualKernel(int n, const float * __restrict__ a,  const float * __restrict__ b, float * __restrict__ c)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i < n)
		c[i] = (llroundf(a[i]) == llroundf (b[i]) ? 1.0 : 0.0);
}


Equal::Equal(deepflow::NodeParam *param) : Node(param) {
	LOG_IF(FATAL, param->has_equal_param() == false) << "param.has_equal_param() == false";
}

void Equal::init() {
	LOG_IF(FATAL, _inputs[0]->value()->size() != _inputs[0]->value()->size()) << "Size mismatch [FAILED]";
	_outputs[0]->initValue(_inputs[0]->value()->dims());	
}

void Equal::forward() {
	auto size = _inputs[0]->value()->size();
	EqualKernel << < numOfBlocks(size), maxThreadsPerBlock >> >(size, _inputs[0]->value()->gpu_data(DF_LINE), _inputs[1]->value()->gpu_data(DF_LINE), (float*)_outputs[0]->value()->gpu_data(DF_LINE));
	DF_KERNEL_CHECK();
}

void Equal::backward() {
	
}

std::string Equal::to_cpp() const
{
	std::string cpp = "auto " + _name + " = df.equal(" + _input_name_for_cpp(0) + ", " + _input_name_for_cpp(1) + ", ";
	cpp += "\"" + _name + "\");";	
	return cpp;
}
