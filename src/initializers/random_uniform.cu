#include "hip/hip_runtime.h"
#include "core/common_cu.h"

#include "initializers/random_uniform.h"
#include "core/variable.h"

__global__ void RandKernel(const float min, const float max,const int n, float* dst) {
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	hiprandState_t state;
	hiprand_init(i, 0, 0, &state);
	if (i < n)
		dst[i] = (hiprand_uniform(&state) * (max - min)) + min;
}

RandomUniform::RandomUniform(const InitParam &param) : Initializer(param) {
	LOG_IF(FATAL, param.has_random_uniform_param() == false);
}

void RandomUniform::apply(Variable *variable) {	
	auto size = variable->output(0)->value()->size();
	LOG(INFO) << "Applying random uniform to variable " << variable->name() << " of shape " << variable->output(0)->value()->toString();
	float min = _param.random_uniform_param().min();
	float max = _param.random_uniform_param().max();
	RandKernel << < numOfBlocks(size), maxThreadsPerBlock >> > (min, max, size, (float*)variable->output(0)->value()->mutableData());
	LOG_IF(FATAL, hipPeekAtLastError() != 0);
}