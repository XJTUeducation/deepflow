#include "hip/hip_runtime.h"

#include "nodes/variable.h"
#include "core/initializer.h"

#include <string>
#include <iostream>

#include "core/common_cu.h"

#include <opencv2/opencv.hpp>

#include <glog/logging.h>

__global__
void VariableClampKernel(const int n, float *x, const float min, const float max)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i < n) {
		float val = x[i];
		if (val < min)
			x[i] = min;
		else if (val > max)
			x[i] = max;
	}
}

Variable::Variable(std::shared_ptr<Initializer> initializer, deepflow::NodeParam *param) : Node(param) {
	LOG_IF(FATAL, param->has_variable_param() == false) << "param.has_variable_param() == false";
	_initializer = initializer;			
}

void Variable::initForward() {	
	_initializer->init();
	_outputs[0]->initValue(_initializer->dims());
	LOG(INFO) << "Variable " << _name << " - " << _outputs[0]->value()->shape();
	if (_param->variable_param().has_weights()) {		
		auto weights = _param->variable_param().weights();
		LOG_IF(FATAL, weights.data_size() != _outputs[0]->value()->size()) << "weights.weight_size() != _outputs[0]->value()->size() in " << _name << " - " << weights.data_size() << " != " << _outputs[0]->value()->size();
		DF_NODE_CUDA_CHECK(hipMemcpy(_outputs[0]->value()->mutableData(), weights.data().data(), _outputs[0]->value()->sizeInBytes(), hipMemcpyHostToDevice));
	}
	else if (_initializer->param()->has_init_data()) {		
		auto weights = _initializer->param()->init_data();
		LOG_IF(FATAL, weights.data_size() != _outputs[0]->value()->size()) << "weights.weight_size() != _outputs[0]->value()->size() in " << _name << " - " << weights.data_size() << " != " << _outputs[0]->value()->size();
		DF_NODE_CUDA_CHECK(hipMemcpy(_outputs[0]->value()->mutableData(), weights.data().data(), _outputs[0]->value()->sizeInBytes(), hipMemcpyHostToDevice));
	}
	else {		
		_initializer->apply(this);
		for (int i = 0; i < _outputs[0]->value()->size(); ++i)
			_param->mutable_variable_param()->mutable_init_param()->mutable_init_data()->add_data(0);
		DF_NODE_CUDA_CHECK(hipMemcpy(_param->mutable_variable_param()->mutable_init_param()->mutable_init_data()->mutable_data()->mutable_data(),_outputs[0]->value()->data(),_outputs[0]->value()->sizeInBytes(), hipMemcpyDeviceToHost));
	}
}

void Variable::initBackward() {	
	_outputs[0]->initDiff();
	int size = _outputs[0]->value()->sizeInBytes();
	DF_CUDA_CHECK(hipMalloc(&_grad, size));
	DF_CUDA_CHECK(hipMemset(_grad, 0, size));
}

inline void Variable::forward() {
	
}

inline void Variable::backward() {
	cpy(_outputs[0]->value()->size(), 1.0, _outputs[0]->diff()->data(), 1.0, _grad);
}

float * Variable::gradients()
{
	return _grad;
}

void Variable::reset_gradients()
{	
	DF_CUDA_CHECK(hipMemset(_grad, 0, _outputs[0]->value()->sizeInBytes()));
}

void Variable::prep_for_saving()
{	
	auto var_param = _param->mutable_variable_param();
	auto mutable_weights = var_param->mutable_weights();
	auto mutable_weights_data = mutable_weights->mutable_data();
	mutable_weights_data->Resize(_outputs[0]->value()->size(),0.0f);
	LOG_IF(FATAL, mutable_weights_data->size() != _outputs[0]->value()->size());
	DF_NODE_CUDA_CHECK(hipMemcpy(mutable_weights_data->mutable_data(), _outputs[0]->value()->data(), _outputs[0]->value()->sizeInBytes(), hipMemcpyDeviceToHost));
}

void Variable::clamp(float min, float max)
{
	auto size = _outputs[0]->value()->size();
	VariableClampKernel << < numOfBlocks(size), maxThreadsPerBlock >> > (size, (float*) _outputs[0]->value()->mutableData(), min, max);
	DF_KERNEL_CHECK();
}

Node::BackwardType Variable::backwardType()
{
	return _param->mutable_variable_param()->solver_name().empty() ? NEVER_BACKWARD : ALWAYS_BACKWARD;
}

std::string Variable::to_cpp() const
{	
	std::string cpp = "auto " + _name + " = df.variable(" + _initializer->to_cpp() + ", ";	
	if (_param->variable_param().solver_name().empty())
		cpp += "NULL, ";
	else
		cpp += _param->variable_param().solver_name() + ", ";
	cpp += "\"" + _name + "\", ";
	cpp += "{" + _to_cpp_phases() + "});";
	return cpp;
}