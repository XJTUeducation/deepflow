#include "hip/hip_runtime.h"
#include "nodes/psnr.h"
#include "core/common_cu.h"

__global__
void SquareErrorKernel(const int n, const float * __restrict__ a, const float * __restrict__ b, float * __restrict__ c)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i < n) {
		float tmp = a[i] - b[i];
		c[i] = tmp * tmp;
	}
}

Psnr::Psnr(deepflow::NodeParam *param) : Node(param) {
	LOG_IF(FATAL, param->has_psnr_param() == false) << "param.has_psnr_param() == false";
	auto psnrParam = _param->psnr_param();	
}

void Psnr::init() {	
	LOG_IF(FATAL, _inputs[0]->value()->size() != _inputs[1]->value()->size()) << "Input " << _inputs[0]->value()->shape() << " != " << " Target " << _inputs[1]->value()->shape();	
	DF_NODE_CUDNN_CHECK(hipdnnCreate(&_cudnnHandle));
	DF_NODE_CUDA_CHECK(hipMalloc(&d_square_error, _inputs[0]->value()->bytes()));
	DF_NODE_CUDA_CHECK(hipMalloc(&d_sum_square_error, sizeof(float)));
	DF_NODE_CUDNN_CHECK(hipdnnCreateReduceTensorDescriptor(&_reduce_tensor_desciptor));
	hipdnnReduceTensorOp_t op = HIPDNN_REDUCE_TENSOR_ADD;
	hipdnnReduceTensorIndices_t indices = HIPDNN_REDUCE_TENSOR_NO_INDICES;
	DF_NODE_CUDNN_CHECK(hipdnnSetReduceTensorDescriptor(_reduce_tensor_desciptor, op, HIPDNN_DATA_FLOAT, HIPDNN_PROPAGATE_NAN, indices, HIPDNN_32BIT_INDICES));	
	DF_NODE_CUDNN_CHECK(hipdnnCreateTensorDescriptor(&_output_desc));
	DF_NODE_CUDNN_CHECK(hipdnnSetTensor4dDescriptor(_output_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, 1, 1, 1));
	DF_NODE_CUDNN_CHECK(hipdnnGetReductionWorkspaceSize(_cudnnHandle, _reduce_tensor_desciptor, _inputs[0]->value()->descriptor(), _output_desc, &_workspaceSizeInBytes));
	DF_NODE_CUDA_CHECK(hipMalloc(&_d_workspace, _workspaceSizeInBytes));
}

void Psnr::forward() {
	auto size = _inputs[0]->value()->size();
	SquareErrorKernel <<< numOfBlocks(size), maxThreadsPerBlock >>> (size, _inputs[0]->value()->gpu_data(), _inputs[1]->value()->gpu_data(), d_square_error);
	DF_KERNEL_CHECK();	
	DF_NODE_CUDNN_CHECK(
		hipdnnReduceTensor(
			_cudnnHandle,
			_reduce_tensor_desciptor,
			NULL,
			NULL,
			_d_workspace,
			_workspaceSizeInBytes,
			&one,
			_inputs[0]->value()->descriptor(),
			d_square_error,
			&zero,
			_output_desc,
			d_sum_square_error));
	float sse;
	DF_NODE_CUDA_CHECK(hipMemcpy(&sse, d_sum_square_error, sizeof(float), hipMemcpyDeviceToHost));
	_psnr = 20.0f * log10(2.0f) - 10.0f * log10(sse / size);	
}

void Psnr::backward() {
}

std::string Psnr::to_cpp() const
{	
	std::string cpp = "df.psnr(" + _input_name_for_cpp(0) + ", " + _input_name_for_cpp(1) + ", ";
	cpp += "\"" + _name + "\");";	
	return cpp;
}
