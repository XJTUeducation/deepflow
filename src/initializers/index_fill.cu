#include "hip/hip_runtime.h"
#include "core/common_cu.h"

#include "initializers/index_fill.h"
#include "nodes/variable.h"

__global__
void IndexFillKernel(const int n, float *a,const float offset)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i < n) a[i] = offset + i;
}

IndexFill::IndexFill(const InitParam &param) : Initializer(param) {
	LOG_IF(FATAL, param.has_index_fill_param() == false) << "param.has_index_fill_param() == false";	
}

void IndexFill::apply(Variable *variable) {
	float offset = _param.index_fill_param().offset();
	auto size = variable->output(0)->value()->size();	
	IndexFillKernel << <numOfBlocks(size), maxThreadsPerBlock >> >(size, (float*)variable->output(0)->value()->mutableData(), offset);
	DF_KERNEL_CHECK();
}

std::string IndexFill::to_cpp() const
{
	std::string cpp = "df.index_fill(";
	cpp += "{" + std::to_string(_dims[0]) + ", " + std::to_string(_dims[1]) + ", " + std::to_string(_dims[2]) + ", " + std::to_string(_dims[3]) + "}, ";
	float offset = _param.index_fill_param().offset();
	cpp += std::to_string(offset) + ")";
	return cpp;
}
