#include "hip/hip_runtime.h"
#include "ops/display.h"
#include "core/common_cu.h"

#include <opencv2/opencv.hpp>
//#include <opencv2/gpu/gpu.hpp>
//#include <opencv2/gpu/gpumat.hpp>

__global__
void PictureGeneratorKernel(const int n, const float *in, int *out, const int picWidth, const int sq, const int numImages, const int perImageWidth, const int perImageHeight)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i < n) {
		int flat_pixel = i % (perImageWidth * perImageHeight);
		int num_image = (i - flat_pixel) / (perImageWidth * perImageHeight);
		int csmall = flat_pixel % perImageWidth;
		int rsmall = (flat_pixel - csmall) / perImageWidth;
		int cbig = num_image % sq;
		int rbig = (num_image - cbig) / sq;
		int c = cbig*perImageWidth + csmall;
		int r = rbig*perImageHeight + rsmall;
		out[r*picWidth + c] = (in[i] + 1.0f) / 2.0f * 255;
	}
}

Display::Display(const NodeParam &param) : Node(param) {
	LOG_IF(FATAL, param.has_display_param() == false) << "param.has_display_param() == false";
}

void Display::initForward() {
	auto dims = _inputs[0]->value()->dims();	
	input_size = _inputs[0]->value()->size();
	input_size_in_bytes = _inputs[0]->value()->sizeInBytes();
	num_images = dims[0];
	per_image_height = dims[2];
	per_image_width = dims[3];
	num_image_per_row_and_col = (int)floor(sqrt((float)num_images));
	pic_width = per_image_width * num_image_per_row_and_col;
	pic_height = per_image_height * ((int)ceil(((float)num_images / num_image_per_row_and_col)));
	num_pic_pixels = pic_width * pic_height;	

	LOG_IF(FATAL, hipMalloc(&d_pic, sizeof(int) * num_pic_pixels) != 0);
	LOG_IF(FATAL, hipMemset(d_pic, 0, sizeof(int) * num_pic_pixels) != 0);
		
	//cv::gpu::GpuMat picMat(pic_height, pic_width, CV_32SC1, img);

	_outputs[0]->initValue(_inputs[0]->value()->dims());
	LOG(INFO) << "Initializing Display " << _name << " - " << pic_width << "x" << pic_height;
	h_pic = new int[num_pic_pixels];
	disp = cv::Mat(pic_height, pic_width, CV_8U, h_pic);	
}

void Display::initBackward() {
	_outputs[0]->initDiff();
}

void Display::forward() {	
	PictureGeneratorKernel << < numOfBlocks(input_size), maxThreadsPerBlock >> >(input_size, (float*) _inputs[0]->value()->data(), d_pic, pic_width, num_image_per_row_and_col, num_images, per_image_width, per_image_height);
	LOG_IF(FATAL, hipPeekAtLastError() != 0);	
	LOG_IF(FATAL, hipMemcpy(h_pic, d_pic, sizeof(int) *num_pic_pixels, hipMemcpyDeviceToHost) != 0);
	for (int r = 0; r < pic_height; r++)
		for (int c = 0; c < pic_width; c++)
			disp.at<char>(r, c) = h_pic[r * pic_width + c];	
	cv::imshow(name(), disp);
	cv::waitKey(100);
	LOG_IF(FATAL,hipMemcpy(_outputs[0]->value()->mutableData(), _inputs[0]->value()->data(), input_size_in_bytes, hipMemcpyDeviceToDevice));
}

void Display::backward() {
	LOG_IF(FATAL, hipMemcpy(_inputs[0]->diff()->mutableData(), _outputs[0]->diff()->data(), input_size_in_bytes, hipMemcpyDeviceToDevice));
}