#include "hip/hip_runtime.h"
#include "ops/square.h"
#include "core/common_cu.h"

__global__
void SquareKernelForward(const int n, const float * __restrict__ x, float * __restrict__ x2)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i < n) x2[i] = x[i] * x[i];
}

__global__
void SquareKernelBackward(const int n, const float *x, const float *diff, float *out)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i < n) out[i] = 2.0f * x[i] * diff[i];
}

Square::Square(const NodeParam &param) : Node(param) {
	LOG_IF(FATAL, param.has_square_param() == false);
}

void Square::initForward() {		
	_outputs[0]->initValue(_inputs[0]->value()->dims());
	LOG(INFO) << "Initializing Square (name: " << _name << " ) | Shape : " << _outputs[0]->value()->shape();
}

void Square::initBackward() {
	_outputs[0]->initDiff();
}

void Square::forward() {	
	auto size = _inputs[0]->value()->size();
	SquareKernelForward <<< numOfBlocks(size), maxThreadsPerBlock >>> (size, (float*)_inputs[0]->value()->data(), (float*)_outputs[0]->value()->mutableData());
	LOG_IF(FATAL, hipPeekAtLastError() != 0);
}

void Square::backward() {	
	auto size = _inputs[0]->value()->size();
	SquareKernelBackward << < numOfBlocks(size), maxThreadsPerBlock >> > (size, (float*)_inputs[0]->value()->data(), (float*)_outputs[0]->diff()->data(), (float*)_inputs[0]->diff()->mutableData());
	LOG_IF(FATAL, hipPeekAtLastError() != 0);	
}