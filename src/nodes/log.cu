#include "hip/hip_runtime.h"
#include "nodes/log.h"
#include "core/common_cu.h"

__global__
void LogKernelForward(const int n, const float coef, const float * __restrict__ x, float * __restrict__ x2)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i < n)
		x2[i] = coef * log(x[i]);
}

__global__
void LogKernelBackward(const int n, const float coef, const float *x, const float *diff, float *out)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i < n)
		out[i] = coef * diff[i] / x[i];
}

Log::Log(deepflow::NodeParam *param) : Node(param) {
	LOG_IF(FATAL, param->has_log_param() == false) << "param.has_log_param() == false";
}

void Log::init() {
	_outputs[0]->initValue(_inputs[0]->value()->dims());
	_outputs[0]->initDiff();
	LOG(INFO) << "Log " << _name << " - " << _outputs[0]->value()->shape();
}

void Log::forward() {
	auto size = _inputs[0]->value()->size();
	auto coef = _param->log_param().coef();
	LogKernelForward << < numOfBlocks(size), maxThreadsPerBlock >> > (size, coef, (float*)_inputs[0]->value()->data(), (float*)_outputs[0]->value()->mutableData());
	DF_KERNEL_CHECK();
}

void Log::backward() {
	auto size = _inputs[0]->value()->size();
	auto coef = _param->log_param().coef();
	LogKernelBackward << < numOfBlocks(size), maxThreadsPerBlock >> > (size, coef, (float*)_inputs[0]->value()->data(), (float*)_outputs[0]->diff()->data(), (float*)_inputs[0]->diff()->mutableData());
	DF_KERNEL_CHECK();
}

std::string Log::to_cpp() const
{
	std::string cpp = "auto " + _name + " = df.log(" + _input_name_for_cpp(0) + ", ";
	cpp += "\"" + _name + "\", ";
	cpp += "{" + _to_cpp_phases() + "});";
	return cpp;
}
