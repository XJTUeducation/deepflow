#include "hip/hip_runtime.h"
#include "core/common_cu.h"

#include "nodes/leaky_relu.h"

__global__
void ReluKernel(int n, const float * __restrict__ x, const float * __restrict__ y, float beta, float * __restrict__ z, const float slope)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i < n)
	{
		if (x[i] > 0)
			z[i] = beta * z[i] + y[i];
		else
			z[i] = beta * z[i] + y[i] * slope;
	}
}

LeakyRelu::LeakyRelu(const deepflow::NodeParam &param) : Node(param) {
	LOG_IF(FATAL, param.has_leaky_relu_param() == false) << "param.has_leaky_relu_param() == false";	
}

void LeakyRelu::initForward() {	
	_negative_slope = _param.leaky_relu_param().negative_slope();
	LOG_IF(FATAL, _negative_slope < 0) << " negative_slope < 0";
	_outputs[0]->initValue(_inputs[0]->value()->dims());
	LOG(INFO) << "Initializing LeakyRelu " << _name << " - " << _outputs[0]->value()->shape();
}

void LeakyRelu::initBackward() {
	_outputs[0]->initDiff();
}

void LeakyRelu::forward() {	
	auto size = _inputs[0]->value()->size();	
	ReluKernel << < numOfBlocks(size), maxThreadsPerBlock >> >(size, (float*)_inputs[0]->value()->data(), (float*)_inputs[0]->value()->data(), 0.0f, (float*)_outputs[0]->value()->mutableData(), _negative_slope);
	DF_KERNEL_CHECK();	
}

void LeakyRelu::backward() {	
	auto size = _inputs[0]->value()->size();	
	ReluKernel << < numOfBlocks(size), maxThreadsPerBlock >> >(size, (float*)_inputs[0]->value()->data(), (float*)_outputs[0]->diff()->data(), 1.0f, (float*)_inputs[0]->diff()->mutableData(), _negative_slope);
	DF_KERNEL_CHECK();	
}

std::string LeakyRelu::to_cpp() const
{
	std::string cpp = "auto " + _name + " = df.leaky_relu(" + _input_name_for_cpp(0) + ", ";
	cpp += std::to_string(_negative_slope) + ", ";
	cpp += "\"" + _name + "\", ";
	cpp += "{" + _to_cpp_phases() + "});";
	return cpp;
}
