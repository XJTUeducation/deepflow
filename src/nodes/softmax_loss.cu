#include "hip/hip_runtime.h"
#include "core/common_cu.h"

#include "nodes/softmax_loss.h"

__global__
void SoftmaxLossKernelBackward(const int n, const float *softmax_output,const float * __restrict__ target, float * __restrict__ diff)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i < n) diff[i] = target[i] - softmax_output[i];
}


SoftmaxLoss::SoftmaxLoss(const deepflow::NodeParam &param) : Loss(param) {
	LOG_IF(FATAL, param.loss_param().has_softmax_loss_param() == false) << "param.loss_param().has_softmax_loss_param() == false";
}

void SoftmaxLoss::initForward() {		
	LOG(INFO) << "Initializing SoftmaxLoss " << _name << " - " << _inputs[0]->value()->shape();
	LOG_IF(FATAL, _inputs[0]->value()->size() != _inputs[1]->value()->size()) << "Input size != target size";
	DF_CUDNN_CHECK(hipdnnCreate(&_cudnnHandle));	
	_outputs[0]->initValue(_inputs[0]->value()->dims());
	_outputs[1]->initValue(_inputs[0]->value()->dims());
}

void SoftmaxLoss::initBackward() {
	_outputs[0]->initDiff();
	_outputs[1]->initDiff();
}

void SoftmaxLoss::forward() {
	DF_CUDNN_CHECK(hipdnnSoftmaxForward(_cudnnHandle, HIPDNN_SOFTMAX_ACCURATE, HIPDNN_SOFTMAX_MODE_CHANNEL, &alpha, _inputs[0]->value()->descriptor(), _inputs[0]->value()->data(), &beta, _outputs[0]->value()->descriptor(), _outputs[1]->value()->mutableData()));
}

void SoftmaxLoss::backward() {
	size_t size = _outputs[0]->value()->size();
	SoftmaxLossKernelBackward << < numOfBlocks(size), maxThreadsPerBlock >> > (size, (float*) _outputs[1]->value()->data(), (float*) _inputs[1]->value()->data(), (float*)_outputs[0]->value()->mutableData());
	DF_KERNEL_CHECK();
	DF_CUDA_CHECK(hipMemcpy(_inputs[0]->diff()->mutableData(), _outputs[0]->value()->data(), _outputs[0]->value()->sizeInBytes(), hipMemcpyDeviceToDevice));	
}

std::string SoftmaxLoss::to_cpp() const
{	
	std::string cpp = "auto " + _name + " = df.softmax_loss(" + _inputs[0]->connectedNode()->name() + ", " + _inputs[1]->connectedNode()->name() + ", ";
	cpp += "\"" + _name + "\", ";
	cpp += "{" + _to_cpp_phases() + "});";
	return cpp;
}
