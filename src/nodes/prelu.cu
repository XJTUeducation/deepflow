#include "hip/hip_runtime.h"
#include "core/common_cu.h"

#include "nodes/prelu.h"

__global__
void PReluForwardKernel(int n, int channels, int inner_dims, const float * __restrict__ x, const float * __restrict__ w, float * __restrict__ y)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;	
	if (i < n)
	{
		if (x[i] > 0) {
			y[i] = x[i];
		}
		else 
		{
			const int iw = (i / inner_dims) % channels;
			y[i] = x[i] * w[iw];
		}
	}
}

__global__
void PReluBackwardKernel(int n, int channels, int inner_dims, const float *x, const float *w, const float * dy, float *dx)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i < n)
	{
		if (x[i] > 0)
			dx[i] = dy[i];
		else {
			const int iw = (i / inner_dims) % channels;
			dx[i] = dy[i] * w[iw];
		}
	}
}

__global__
void PReluBackwardWeightKernel(int n, int channels, int inner_dims, const float *x, const float *w, const float * dy, float *dw)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i < n && x[i] < 0)
	{
		int iw = (i / inner_dims) % channels;		
		atomicAdd(&dw[iw], dy[i] * x[i] * channels / n);
	}
}

PRelu::PRelu(deepflow::NodeParam * param) : Node(param)
{
	LOG_IF(FATAL, param->has_prelu_param() == false) << "param.has_prelu_param() == false";
}

void PRelu::init()
{
	auto wdims = _inputs[1]->dims();
	auto idims = _inputs[0]->dims();
	LOG_IF(FATAL, wdims[0] != 1 || wdims[1] != idims[1] || wdims[2] != 1 || wdims[3] != 1) << "PRELU " << _name << " weights must be " << "1x" << idims[1] << "x1x1";
	_outputs[0]->initValue(idims);
	_outputs[0]->initDiff();	
}

void PRelu::forward()
{
	auto size = _inputs[0]->value()->size();
	auto channels = _inputs[0]->dims()[1];
	auto inner_dims = _inputs[0]->dims()[2] * _inputs[0]->dims()[3];
	PReluForwardKernel << < numOfBlocks(size), maxThreadsPerBlock>> >(size, channels, inner_dims, (float*)_inputs[0]->value()->data(), (float*)_inputs[1]->value()->data(), (float*)_outputs[0]->value()->mutableData());
	DF_NODE_KERNEL_CHECK();
}

void PRelu::backward()
{
	auto size = _inputs[0]->value()->size();
	auto channels = _inputs[0]->dims()[1];
	auto inner_dims = _inputs[0]->dims()[2] * _inputs[0]->dims()[3];
	PReluBackwardKernel << < numOfBlocks(size), maxThreadsPerBlock>> >(size, channels, inner_dims, (float*)_inputs[0]->value()->data(), (float*)_inputs[1]->value()->data(), (float*)_outputs[0]->diff()->data(), (float*)_inputs[0]->diff()->mutableData());
	DF_NODE_KERNEL_CHECK();
	DF_CUDA_CHECK(hipMemset(_inputs[1]->diff()->mutableData(), 0, _inputs[1]->diff()->sizeInBytes()));
	PReluBackwardWeightKernel << < numOfBlocks(size), maxThreadsPerBlock>> >(size, channels, inner_dims, (float*)_inputs[0]->value()->data(), (float*)_inputs[1]->value()->data(), (float*)_outputs[0]->diff()->data(), (float*)_inputs[1]->diff()->mutableData());
	DF_NODE_KERNEL_CHECK();
}

std::string PRelu::to_cpp() const
{	
	std::string cpp = "auto " + _name + " = df.prelu(" + _input_name_for_cpp(0) + ", " + _input_name_for_cpp(1) + ", ";
	cpp += "\"" + _name + "\", ";
	cpp += "{" + _to_cpp_phases() + "});";
	return cpp;
}
