#include "hip/hip_runtime.h"
#include "nodes/accumulator.h"

#include "core/common_cu.h"

__global__
void AccumulatorKernel(int n, const float * __restrict__ x, float * __restrict__ out1)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i < n)
		out1[i] += x[i];
}

Accumulator::Accumulator(deepflow::NodeParam *param) : Node(param) {
	LOG_IF(FATAL, param->has_accumulator_param() == false) << "param.has_accumulator_param() == false";
}

void Accumulator::init() {	
	const deepflow::AccumulatorParam &accParam = _param->accumulator_param();
	_reset_time = (deepflow::ActionTime)accParam.reset_time();
	_outputs[0]->initValue(_inputs[0]->value()->dims());
	_outputs[1]->initValue({ 1,1,1,1 });	
}

void Accumulator::forward() {
	if (_reset_time == deepflow::ActionTime::END_OF_EPOCH && _context->current_iteration_per_epoch == 1) {
		DF_NODE_CUDA_CHECK(hipMemset(_outputs[0]->value()->mutableData(), 0, _outputs[0]->value()->sizeInBytes()));
		_total = 0;		
	}
	auto size = _inputs[0]->value()->size();
	AccumulatorKernel << < numOfBlocks(size), maxThreadsPerBlock >> >(size, (float*)_inputs[0]->value()->data(), (float*)_outputs[0]->value()->mutableData());
	DF_KERNEL_CHECK();
	_total += _inputs[0]->dims()[0];	
	DF_NODE_CUDA_CHECK(hipMemcpy(_outputs[1]->value()->mutableData(), &_total, sizeof(float), hipMemcpyHostToDevice));
}

void Accumulator::backward() {
	LOG(FATAL);
}

std::string Accumulator::to_cpp() const
{
	std::string cpp = "auto " + _name + " = df.accumulator(" + _input_name_for_cpp(0) + ", ";
	if (_reset_time == deepflow::ActionTime::END_OF_EPOCH)
		cpp += "Accumulator::EndOfEpoch, ";
	else if (_reset_time == deepflow::ActionTime::NEVER)
		cpp += "Accumulator::Never, ";
	cpp += "\"" + _name + "\", ";
	cpp += "{" + _to_cpp_phases() + "});";
	return cpp;
}
