#include "hip/hip_runtime.h"
#include "solvers/gain_solver.h"
#include "core/common_cu.h"
#include "core/variable.h"

#include "core/reader.h"
#include "observers/forward.h"
#include "observers/backward.h"
#include "observers/reset.h"

#include <glog/logging.h>

__global__
void GainFillKernel(int n, float *a)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i < n) a[i] = 1.0f;
}

__global__
void GainStepKernel(const int n, float *current_weight, float *current_gradient, float *previous_gradient, float *gain, const float max_gain, const float min_gain, const float gain_plus, const float gain_mult, const float momentum, const float learning_rate)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i < n) {
		if (previous_gradient[i] * current_gradient[i] > 0)
			gain[i] += gain_plus;
		else
			gain[i] *= gain_mult;
		if (gain[i] > max_gain)
			gain[i] = max_gain;
		if (gain[i] < min_gain)
			gain[i] = min_gain;
		previous_gradient[i] = current_gradient[i];
		current_gradient[i] *= gain[i];
		current_weight[i] = momentum * current_weight[i] + learning_rate * current_gradient[i];
	}
}

GainSolver::GainSolver(NodeOutputPtr loss, const SolverParam &param) : Solver(loss,param) {
	LOG_IF(FATAL, param.has_gain_solver() == false) << "param.has_gain_solver() == false";
	_my_param = param.gain_solver();
}

void GainSolver::train_step() {
	if (_initialized == false) {
		init();
	}	
	ResetObserver resetObserver;
	ForwardObserver forwardObserver;
	BackwardObserver backwardObserver;
	_loss_node->traverse(&resetObserver, TraverseOrder::PreOrder, true);
	_loss_node->traverse(&forwardObserver, TraverseOrder::PostOrder, false);
	_loss_node->traverse(&resetObserver, TraverseOrder::PreOrder, true);
	_loss_node->traverse(&backwardObserver, TraverseOrder::PreOrder, false);	
	int index = 0;	
	for( auto var : _variables)
	{		
		auto output = var->output(0);
		auto size = output->value()->size();
		GainStepKernel << <numOfBlocks(size), maxThreadsPerBlock, 0 , _streams[index]>> > (size, (float*) output->value()->mutableData(), (float*) output->diff()->mutableData(), _previous_gradients[index], _gains[index], _my_param.max_gain(), _my_param.min_gain(), _my_param.gain_plus(), _my_param.gain_mult(), _my_param.momentum(), _my_param.learning_rate());
		LOG_IF(FATAL, hipPeekAtLastError() != 0);		
		index++;
	}

	hipDeviceSynchronize();
	
	for (auto var : _variables)
	{		
		if (var->snapshot() && _current_step % var->snapshotInterval() == 0)
			var->toImage(_current_step);
	}

	_current_step++;
}

void GainSolver::init() {
	for (auto var : _variables)
	{
		_streams.push_back(hipStream_t());
		hipStreamCreate(&_streams.back());
		auto size = var->output(0)->value()->size();
		auto sizeInBytes = var->output(0)->value()->sizeInBytes();
		_previous_gradients.push_back(NULL);
		LOG_IF(FATAL, hipMalloc(&_previous_gradients.back(), sizeInBytes) != 0);
		LOG_IF(FATAL, hipMemset(_previous_gradients.back(), 0, sizeInBytes) != 0);
		_gains.push_back(NULL);
		LOG_IF(FATAL, hipMalloc(&_gains.back(), sizeof(float) * size) != 0);
		GainFillKernel <<<numOfBlocks(size), maxThreadsPerBlock>>>(size, _gains.back());
		LOG_IF(FATAL, hipPeekAtLastError() != 0);
	}
	_initialized = true;
}