#include "hip/hip_runtime.h"
#include "generators/image_reader.h"
#include "core/common_cu.h"
#include <opencv2/opencv.hpp>

__global__
void NormalizeKernel(const int n, const unsigned char *in, float *out)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i < n) {
		out[i] = (((float)in[i] / 255.0f) - 0.5f) * 2;
	}
}

__global__
void ConvertOpenCV3ImageKernel(const int n, const unsigned char *in, const int width, const int height, float *out)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i < n) {
		int channel = i % 3;
		int input_pixel = (i - channel) / 3;
		int input_col = input_pixel % width;
		int input_row = (input_pixel - input_col) / width;
		out[ (2 - channel) * width * height + input_row * width + input_col] = (((float)in[i] / 255.0f) - 0.5f) * 2;
	}
}


ImageReader::ImageReader(const deepflow::NodeParam &param) : Node(param), Generator(param) {
	LOG_IF(FATAL, param.generator_param().has_image_reader_param() == false) << "param.generator_param().has_image_reader_param() == false";
}

void ImageReader::nextBatch() {

}

void ImageReader::initForward() {
	auto image_reader_param = _param.generator_param().image_reader_param();
	auto file_name = image_reader_param.file_name();		
	auto type = image_reader_param.type();
	if (type == deepflow::ImageReaderParam_Type_GRAY_ONLY)
		img = cv::imread(file_name, 0);
	else
		img = cv::imread(file_name);
	LOG_IF(FATAL, img.empty()) << "Image " << file_name << " does not exist.";
	_outputs[0]->initValue({ 1, img.channels(), img.rows , img.cols });
	LOG(INFO) << "Initializing image_readr for image " << file_name << " - " << _outputs[0]->value()->shape();
	size_t size = _outputs[0]->value()->size();
	unsigned char *d_img;
	DF_CUDA_CHECK(hipMalloc(&d_img, size));
	DF_CUDA_CHECK(hipMemcpy(d_img, img.ptr<uchar>(), size, hipMemcpyHostToDevice));
	if (img.channels() == 1) {
		NormalizeKernel << < numOfBlocks(size), maxThreadsPerBlock >> > (size, d_img, (float*)_outputs[0]->value()->mutableData());
		DF_KERNEL_CHECK();
	}
	else if (img.channels() == 3) {
		ConvertOpenCV3ImageKernel << < numOfBlocks(size), maxThreadsPerBlock >> > (size, d_img, img.cols, img.rows, (float*)_outputs[0]->value()->mutableData());
		DF_KERNEL_CHECK();
	}
	else {
		LOG(FATAL) << "Unsupported image.";
	}
	DF_CUDA_CHECK(hipFree(d_img));
	_last_batch = true;
}

bool ImageReader::isLastBatch() {
	return _last_batch;
}

std::string ImageReader::to_cpp() const
{
	auto image_reader_param = _param.generator_param().image_reader_param();
	auto file_name = image_reader_param.file_name();
	auto type = image_reader_param.type();	
	std::string cpp = "auto " + _name + " = df.image_reader(\"" + file_name + "\", ";
	if (type == deepflow::ImageReaderParam_Type_GRAY_ONLY)
		cpp += "ImageReaderParam_Type_GRAY_ONLY, ";
	else
		cpp += "ImageReaderParam_Type_COLOR_IF_AVAILABLE, ";
	cpp += "\"" + _name + "\", ";
	cpp += "{" + _to_cpp_phases() + "});";
	return cpp;
}
