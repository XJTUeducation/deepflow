#include "hip/hip_runtime.h"
#include "solvers/adadelta_solver.h"
#include "core/common_cu.h"
#include "core/variable.h"

#include <glog/logging.h>

__global__
void FillKernel(int n, float *a)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i < n) a[i] = 1.0f;
}

__global__
void AdaDeltaKernel(const int n, float *w, const float *g, float *h1, float *h2, const float momentum, const float learning_rate, const float delta)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i < n) {
		float gi = g[i];
		float hi = h1[i] = momentum * h1[i] + (1 - momentum) * gi * gi;
		gi = gi * sqrt((h2[i] + delta) / (hi + delta));
		h2[i] = momentum * h2[i] + (1 - momentum) * gi * gi;
		w[i] += learning_rate * gi;
	}
}

AdaDeltaSolver::AdaDeltaSolver(const SolverParam &param) : Solver(param) {
	LOG_IF(FATAL, param.has_adadelta_solver() == false) << "param.has_adadelta_solver() == false";
	_my_param = param.adadelta_solver();
}

void AdaDeltaSolver::apply(std::shared_ptr<Variable> var) {
	if (_initialized == false)
		init(var);
	auto output = var->output(0);
	auto size = output->value()->size();
	AdaDeltaKernel << <numOfBlocks(size), maxThreadsPerBlock >> > (size, (float*)output->value()->mutableData(), (float*)output->diff()->data(), _h1, _h2, _my_param.momentum(), _my_param.learning_rate(), _my_param.delta());
	DF_KERNEL_CHECK();
}

void AdaDeltaSolver::init(std::shared_ptr<Variable> var) {
	auto size = var->output(0)->value()->size();
	auto sizeInBytes = var->output(0)->value()->sizeInBytes();
	DF_CUDA_CHECK(hipMalloc(&_h1, sizeInBytes));	
	FillKernel << <numOfBlocks(size), maxThreadsPerBlock >> >(size, _h1);
	DF_KERNEL_CHECK();
	DF_CUDA_CHECK(hipMalloc(&_h2, sizeInBytes));	
	FillKernel << <numOfBlocks(size), maxThreadsPerBlock >> >(size, _h2);
	DF_KERNEL_CHECK();
	_initialized = true;
}