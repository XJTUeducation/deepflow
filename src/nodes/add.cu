#include "hip/hip_runtime.h"
#include "core/common_cu.h"

#include "nodes/add.h"

__global__
void AddKernelForward(const int n, const float alpha, const float *a, const float beta, const float *b, float *c)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i < n) c[i] = alpha * a[i] + beta * b[i];
}

__global__
void AddKernelBackward(const int n, const float *dy, const float scale, float * __restrict__ dx)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i < n) {
		dx[i] = scale * dy[i];		
	}
}

Add::Add(const deepflow::NodeParam &param) : Node(param) {
	LOG_IF(FATAL, param.has_add_param() == false) << "param.has_add_param() == false";	
}

void Add::initForward() {
	
	auto a = _inputs[0];
	auto ad = a->dims();

	auto b = _inputs[1];
	auto bd = b->dims();

	_alpha = _param.add_param().alpha();
	_beta = _param.add_param().beta();

	LOG_IF(FATAL, a->value()->size() != b->value()->size()) << "Different input sizes";		
	_outputs[0]->initValue(_inputs[0]->value()->dims());
	LOG(INFO) << "Initializing Add " << _name << " - " << _outputs[0]->value()->shape();
}

void Add::initBackward() {
	_outputs[0]->initDiff();
}

void Add::forward() {	
	// C(m,n) = A(m,n) + B(m,n)	
	auto size = _outputs[0]->value()->size();
	AddKernelForward << <numOfBlocks(size), maxThreadsPerBlock >> >(size , _alpha, (float*)_inputs[0]->value()->data(),_beta, (float*)_inputs[1]->value()->data(), (float*)_outputs[0]->value()->mutableData());
	DF_KERNEL_CHECK();
}

void Add::backward() {	
	auto size = _outputs[0]->diff()->size();
	if (_inputs[0]->connectedNode()->shouldBackward()) {
		AddKernelBackward << <numOfBlocks(size), maxThreadsPerBlock >> > (size, (float*)_outputs[0]->diff()->data(), _alpha, (float*)_inputs[0]->diff()->mutableData());
		DF_KERNEL_CHECK();
	}	
	if (_inputs[1]->connectedNode()->shouldBackward()) {
		AddKernelBackward << <numOfBlocks(size), maxThreadsPerBlock >> > (size, (float*)_outputs[0]->diff()->data(), _beta, (float*)_inputs[1]->diff()->mutableData());
		DF_KERNEL_CHECK();
	}
}

std::string Add::to_cpp() const
{
	std::string op;
	float print_alpha_beta = false;
	if (_alpha == 1 && _beta == 1)
		op = "add";
	else if (_alpha == 1 && _beta == -1)
		op = "subtract";
	else {
		op = "add";
		print_alpha_beta = true;
	}
	std::string cpp = "auto " + _name + " = df." + op + "(" + _inputs[0]->connectedNode()->name() + ", " + _inputs[1]->connectedNode()->name() + ", ";
	if (print_alpha_beta)
		cpp += std::to_string(_alpha) + ", " + std::to_string(_beta);
	cpp += "\"" + _name + "\", ";
	cpp += "{" + _to_cpp_phases() + "});";
	return cpp;
}
