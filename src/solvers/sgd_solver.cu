#include "hip/hip_runtime.h"
#include "solvers/sgd_solver.h"
#include "core/common_cu.h"

#include <algorithm>

#include "nodes/variable.h"

__global__
void ApplyGradientKernel(const int n, const float momentum, const float learning_rate, float *w, const float *g, float *h)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i < n) {
		float gi = h[i] = momentum*h[i] + learning_rate*g[i];
		w[i] -= gi;
	}
}

SGDSolver::SGDSolver(deepflow::SolverParam *param) : Solver(param) {
	LOG_IF(FATAL, param->has_sgd_solver() == false) << "param.has_sgd_solver() == false";
	_my_param = _param->mutable_sgd_solver();
	_learning_rate = param->learning_rate();
}

void SGDSolver::apply(std::shared_ptr<Variable> var) {
	auto context = var->executionContext();
	bool verbos = (context && context->debug_level > 3) ? true : false;
	if (_initialized == false) {
		LOG_IF(INFO, verbos) << "solver " << name() << " for variable " << var->name();
		init(var);
	}
	if (!_enabled)
		return;
	LOG_IF(INFO, verbos) << "applying solver " << name() << " on " << var->name();
	auto size = var->output(0)->value()->size();
	ApplyGradientKernel << <numOfBlocks(size), maxThreadsPerBlock, 0>> > (size, _my_param->momentum(), _learning_rate, (float*)var->output(0)->value()->gpu_data(), (float*)var->gradients(), _h);
	DF_KERNEL_CHECK();
	var->reset_gradients();
}

void SGDSolver::init(std::shared_ptr<Variable> var) {
	auto size = var->output(0)->value()->size();
	auto sizeInBytes = var->output(0)->value()->bytes();
	DF_CUDA_CHECK(hipMalloc(&_h, sizeInBytes));
	DF_CUDA_CHECK(hipMemset(_h, 0, sizeInBytes));
	_initialized = true;
}

std::string SGDSolver::to_cpp() const
{
	std::string cpp = "auto " + name() + " = df.sgd_solver(";
	cpp += std::to_string(_my_param->momentum()) + ", ";
	cpp += std::to_string(_param->learning_rate()) + ", ";	
	cpp += "\"" + name() + "\"";
	cpp += ");";
	return cpp;
}
