#include "hip/hip_runtime.h"
#include "core/common_cu.h"

#include "initializers/fill.h"
#include "nodes/variable.h"

__global__
void FillKernel(const int n, float *a, const float v)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i < n) a[i] = v;
}

Fill::Fill(const deepflow::InitParam &_block_param) : Initializer(_block_param) {
	LOG_IF(FATAL, _block_param.has_fill_param() == false) << "param.has_fill_param() == false";		
}

void Fill::apply(Variable *variable) {
	float value = _param.fill_param().value();
	auto size = variable->output(0)->value()->size();	
	FillKernel <<< numOfBlocks(size), maxThreadsPerBlock >>> (size, (float*)variable->output(0)->value()->mutableData(), value);
	DF_KERNEL_CHECK();
}

std::string Fill::to_cpp() const
{	
	float value = _param.fill_param().value();
	std::string op;
	bool omit_value = true;
	if (value == 0)
		op = "zeros";
	else if (value == 1)
		op = "ones";
	else {
		op = "fill";
		omit_value = false;
	}
	std::string cpp = "df."+op+"(";
	cpp += "{" + std::to_string(_dims[0]) + ", " + std::to_string(_dims[1]) + ", " + std::to_string(_dims[2]) + ", " + std::to_string(_dims[3]) + "}, ";
	if (!omit_value)
		cpp += std::to_string(value);
	cpp += ")";
	return cpp;
}
