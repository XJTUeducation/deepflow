
#include "nodes/variable.h"
#include "core/initializer.h"

#include <string>
#include <iostream>

#include "core/common_cu.h"

#include <opencv2/opencv.hpp>

#include <glog/logging.h>

Variable::Variable(std::shared_ptr<Initializer> initializer, const deepflow::NodeParam &_block_param) : Node(_block_param) {
	LOG_IF(FATAL, _block_param.has_variable_param() == false) << "param.has_variable_param() == false";
	_initializer = initializer;			
}

void Variable::initForward() {	
	_initializer->init();
	_outputs[0]->initValue(_initializer->dims());
	LOG(INFO) << "Initializing Variable " << _name << " - " << _outputs[0]->value()->shape();
	if (_param.variable_param().has_weights()) {
		auto weights = _param.variable_param().weights();
		LOG_IF(FATAL, weights.weight_size() != _outputs[0]->value()->size()) << "weights.weight_size() != _outputs[0]->value()->size() in " << _name;
		DF_CUDA_CHECK(hipMemcpy(_outputs[0]->value()->mutableData(), weights.weight().data(), _outputs[0]->value()->sizeInBytes(), hipMemcpyHostToDevice));
	}
	else if (_initializer->_block_param().has_init_data()) {
		auto weights = _initializer->_block_param().init_data();
		LOG_IF(FATAL, weights.weight_size() != _outputs[0]->value()->size()) << "weights.weight_size() != _outputs[0]->value()->size() in " << _name;
		DF_CUDA_CHECK(hipMemcpy(_outputs[0]->value()->mutableData(), weights.weight().data(), _outputs[0]->value()->sizeInBytes(), hipMemcpyHostToDevice));
	}
	else {
		_initializer->apply(this);
		for (int i = 0; i < _outputs[0]->value()->size(); ++i)
			_param.mutable_variable_param()->mutable_init_param()->mutable_init_data()->add_weight(0);
		DF_CUDA_CHECK(hipMemcpy(_param.mutable_variable_param()->mutable_init_param()->mutable_init_data()->mutable_weight()->mutable_data(),_outputs[0]->value()->data(),_outputs[0]->value()->sizeInBytes(), hipMemcpyDeviceToHost));
	}
}

void Variable::initBackward() {	
	_outputs[0]->initDiff();
}

void Variable::transferDataToParam() {
	auto dma = _param.mutable_variable_param()->mutable_weights();
	for (int i = 0; i < _outputs[0]->value()->size(); ++i)
		dma->add_weight(0);
	DF_CUDA_CHECK(hipMemcpy(dma->mutable_weight()->mutable_data(), _outputs[0]->value()->data(), _outputs[0]->value()->sizeInBytes(), hipMemcpyDeviceToHost));
}

std::string Variable::to_cpp() const
{	
	std::string cpp = "auto " + _name + " = df.variable(" + _initializer->to_cpp() + ", ";	
	if (_param.variable_param().solver_name().empty())
		cpp += "NULL, ";
	else
		cpp += _param.variable_param().solver_name() + ", ";
	cpp += "\"" + _name + "\", ";
	cpp += "{" + _to_cpp_phases() + "});";
	return cpp;
}