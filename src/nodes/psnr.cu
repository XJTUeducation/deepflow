#include "hip/hip_runtime.h"
#include "nodes/psnr.h"
#include "core/common_cu.h"

__global__
void SquareErrorKernel(const int n, const float * __restrict__ a, const float * __restrict__ b, float * __restrict__ c)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i < n) {
		float tmp = a[i] - b[i];
		c[i] = tmp * tmp;
	}
}

Psnr::Psnr(deepflow::NodeParam *param) : Node(param) {
	LOG_IF(FATAL, param->has_psnr_param() == false) << "param.has_psnr_param() == false";
	auto psnrParam = _param->psnr_param();	
	_print_time = psnrParam.print_time();	
}

void Psnr::init() {	
	LOG_IF(FATAL, _inputs[0]->value()->size() != _inputs[1]->value()->size()) << "Input " << _inputs[0]->value()->shape() << " != " << " Target " << _inputs[1]->value()->shape();
	LOG(INFO) << "PSNR " << _name;
	DF_NODE_CUDNN_CHECK(hipdnnCreate(&_cudnnHandle));
	DF_NODE_CUDA_CHECK(hipMalloc(&d_square_error, _inputs[0]->value()->sizeInBytes()));
	DF_NODE_CUDA_CHECK(hipMalloc(&d_sum_square_error, sizeof(float)));
	DF_NODE_CUDNN_CHECK(hipdnnCreateReduceTensorDescriptor(&_reduce_tensor_desciptor));
	hipdnnReduceTensorOp_t op = HIPDNN_REDUCE_TENSOR_ADD;
	hipdnnReduceTensorIndices_t indices = HIPDNN_REDUCE_TENSOR_NO_INDICES;
	DF_NODE_CUDNN_CHECK(hipdnnSetReduceTensorDescriptor(_reduce_tensor_desciptor, op, HIPDNN_DATA_FLOAT, HIPDNN_PROPAGATE_NAN, indices, HIPDNN_32BIT_INDICES));	
	DF_NODE_CUDNN_CHECK(hipdnnCreateTensorDescriptor(&_output_desc));
	DF_NODE_CUDNN_CHECK(hipdnnSetTensor4dDescriptor(_output_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, 1, 1, 1));
	DF_NODE_CUDNN_CHECK(hipdnnGetReductionWorkspaceSize(_cudnnHandle, _reduce_tensor_desciptor, _inputs[0]->value()->descriptor(), _output_desc, &_workspaceSizeInBytes));
	DF_NODE_CUDA_CHECK(hipMalloc(&_d_workspace, _workspaceSizeInBytes));
}

void Psnr::forward() {
	if (_print_time == deepflow::ActionTime::END_OF_EPOCH && _context->last_batch == false)
		return;
	auto size = _inputs[0]->value()->size();
	SquareErrorKernel <<< numOfBlocks(size), maxThreadsPerBlock >>> (size, (float*)_inputs[0]->value()->data(), (float*)_inputs[1]->value()->data(), d_square_error);
	DF_KERNEL_CHECK();	
	DF_NODE_CUDNN_CHECK(
		hipdnnReduceTensor(
			_cudnnHandle,
			_reduce_tensor_desciptor,
			NULL,
			NULL,
			_d_workspace,
			_workspaceSizeInBytes,
			&one,
			_inputs[0]->value()->descriptor(),
			d_square_error,
			&zero,
			_output_desc,
			d_sum_square_error));
	float sse;
	DF_NODE_CUDA_CHECK(hipMemcpy(&sse, d_sum_square_error, sizeof(float), hipMemcpyDeviceToHost));
	float psnr = 20.0f * log10(2.0f) - 10.0f * log10(sse / size);
	LOG(INFO) << _name << " - PSNR: " << psnr;
}

void Psnr::backward() {
}

std::string Psnr::to_cpp() const
{	
	std::string cpp = "df.psnr(" + _input_name_for_cpp(0) + ", " + _input_name_for_cpp(1) + ", ";
	if (_print_time == deepflow::ActionTime::END_OF_EPOCH) {
		cpp += "Psnr::END_OF_EPOCH, ";
	}
	else {
		cpp += "Psnr::EVERY_PASS, ";
	}
	cpp += "\"" + _name + "\", ";
	cpp += "{" + _to_cpp_phases() + "});";
	return cpp;
}
