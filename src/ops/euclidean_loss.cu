#include "hip/hip_runtime.h"

#include "core/common_cu.h"

#include "ops/euclidean_loss.h"

__global__
void EuclideanLossKernel(const int n, const float * __restrict__ x1, const float * __restrict__ x2, float * __restrict__ y)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i < n)
		y[i] = x1[i] - x2[i];
}

EuclideanLoss::EuclideanLoss(const NodeParam &param) : Loss(param) {
	LOG_IF(FATAL, param.loss_param().has_euclidean_loss_param() == false) << "param.loss_param().has_euclidean_loss_param() == false";
}

void EuclideanLoss::initForward() {
	LOG(INFO) << "Initializing EuclideanLoss " << _name << " - " << _inputs[0]->value()->shape();
	LOG_IF(FATAL, _inputs[0]->value()->size() != _inputs[1]->value()->size()) << "Input " << _inputs[0]->value()->shape() << " != " << " Target " << _inputs[1]->value()->shape();
	_outputs[0]->initValue(_inputs[0]->value()->dims());	
}

void EuclideanLoss::initBackward() {	
	_outputs[0]->initDiff();	
}

void EuclideanLoss::forward() {
	auto size = _inputs[0]->value()->size();
	EuclideanLossKernel << < numOfBlocks(size), maxThreadsPerBlock >> > (size, (float*)_inputs[0]->value()->data(), (float*)_inputs[1]->value()->data(), (float*)_outputs[0]->value()->mutableData());
	DF_KERNEL_CHECK();
}

void EuclideanLoss::backward() {	
	DF_CUDA_CHECK(hipMemcpy(_inputs[0]->diff()->mutableData(), _outputs[0]->value()->data(), _outputs[0]->value()->sizeInBytes(), hipMemcpyDeviceToDevice));
}
