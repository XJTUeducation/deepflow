#include "hip/hip_runtime.h"
#include "core/common_cu.h"

#include "ops/softmax_loss.h"

#include <glog/logging.h>

__global__
void SoftmaxLossKernelBackward(const int n, const float *softmax_output,const float * __restrict__ target, float * __restrict__ diff)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i < n) diff[i] = target[i] - softmax_output[i];
}


SoftmaxLoss::SoftmaxLoss(NodeParam param ) : Node(param) {
	LOG_IF(FATAL, param.loss_param().has_softmax_loss_param() == false);
}

void SoftmaxLoss::initForward() {		
	LOG(INFO) << "Initializing SoftmaxLoss (name: " << _name << " ) | Shape : " << _inputs[0]->value()->toString();
	LOG_IF(FATAL, _inputs[0]->value()->size() != _inputs[1]->value()->size()) << "Input size != target size";
	LOG_IF(FATAL, hipdnnCreate(&_cudnnHandle) != 0);	
	_outputs[0]->initValue(_inputs[0]->value()->dims());
	_outputs[1]->initValue(_inputs[0]->value()->dims());
}

void SoftmaxLoss::initBackward() {
	_outputs[0]->initDiff();
	_outputs[1]->initDiff();
}

void SoftmaxLoss::forward() {
	LOG_IF(FATAL, hipdnnSoftmaxForward(_cudnnHandle, HIPDNN_SOFTMAX_ACCURATE, HIPDNN_SOFTMAX_MODE_CHANNEL, &alpha, _inputs[0]->value()->descriptor(), _inputs[0]->value()->data(), &beta, _outputs[0]->value()->descriptor(), _outputs[1]->value()->mutableData()) != 0);
	LOG_IF(FATAL, hipPeekAtLastError() != 0);	
}

void SoftmaxLoss::backward() {
	size_t size = _outputs[0]->value()->size();
	SoftmaxLossKernelBackward << < numOfBlocks(size), maxThreadsPerBlock >> > (size, (float*) _outputs[1]->value()->data(), (float*) _inputs[1]->value()->data(), (float*)_outputs[0]->value()->mutableData());
	LOG_IF(FATAL, hipPeekAtLastError() != 0);
	LOG_IF(FATAL, hipMemcpy(_inputs[0]->diff()->mutableData(), _outputs[0]->value()->data(), _outputs[0]->value()->sizeInBytes(), hipMemcpyDeviceToDevice) != 0) << "hipMemcpy [FAILED]";
}