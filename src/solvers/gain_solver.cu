#include "hip/hip_runtime.h"
#include "solvers/gain_solver.h"
#include "core/common_cu.h"
#include "core/variable.h"

#include "core/reader.h"
#include "observers/forward.h"
#include "observers/backward.h"
#include "observers/reset.h"

#include <glog/logging.h>

__global__
void GainFillKernel(int n, float *a)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i < n) a[i] = 1.0f;
}

__global__
void GainStepKernel(const int n, float *current_weight, float *current_gradient, float *previous_gradient, float *gain, const float max_gain, const float min_gain, const float gain_plus, const float gain_mult, const float momentum, const float learning_rate)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i < n) {
		if (previous_gradient[i] * current_gradient[i] > 0)
			gain[i] += gain_plus;
		else
			gain[i] *= gain_mult;
		if (gain[i] > max_gain)
			gain[i] = max_gain;
		if (gain[i] < min_gain)
			gain[i] = min_gain;
		previous_gradient[i] = current_gradient[i];
		current_gradient[i] *= gain[i];
		current_weight[i] = momentum * current_weight[i] + learning_rate * current_gradient[i];
	}
}

GainSolver::GainSolver(const SolverParam &param): Solver(param) {
	LOG_IF(FATAL, param.has_gain_solver() == false) << "param.has_gain_solver() == false";
	_my_param = param.gain_solver();
}

void GainSolver::apply(std::shared_ptr<Variable> var) {
	if (_initialized == false) 
		init(var);
	auto output = var->output(0);
	auto size = output->value()->size();
	GainStepKernel << <numOfBlocks(size), maxThreadsPerBlock>> > (size, (float*) output->value()->mutableData(), (float*) output->diff()->mutableData(), _previous_gradient, _gain, _my_param.max_gain(), _my_param.min_gain(), _my_param.gain_plus(), _my_param.gain_mult(), _my_param.momentum(), _my_param.learning_rate());
	LOG_IF(FATAL, hipPeekAtLastError() != 0);		
}

void GainSolver::init(std::shared_ptr<Variable> var) {
	auto size = var->output(0)->value()->size();
	auto sizeInBytes = var->output(0)->value()->sizeInBytes();		
	LOG_IF(FATAL, hipMalloc(&_previous_gradient, sizeInBytes) != 0);
	LOG_IF(FATAL, hipMemset(_previous_gradient, 0, sizeInBytes) != 0);
	LOG_IF(FATAL, hipMalloc(&_gain, sizeInBytes) != 0);
	GainFillKernel <<<numOfBlocks(size), maxThreadsPerBlock>>>(size, _gain);
	LOG_IF(FATAL, hipPeekAtLastError() != 0);	
	_initialized = true;
}