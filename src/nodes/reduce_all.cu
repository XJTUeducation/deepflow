#include "hip/hip_runtime.h"
#include "nodes/reduce_all.h"
#include "core/common_cu.h"

__global__
void ReduceAllKernelForward(const int n, bool average, const float *x, float *y)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i < n) {		
		atomicAdd(&y[0], average ? x[i] / n : x[i]);
	}
}

__global__
void ReduceAllKernelBackward(const int n, bool average, const float *dY, float *dX)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i < n) {
		dX[i] = (average ? dY[0] / n : dY[0]);
	}
		
}

ReduceAll::ReduceAll(deepflow::NodeParam *param) : Node(param) {
	LOG_IF(FATAL, param->has_reduce_all_param() == false) << "param.has_reduce_all_param() == false";
}

void ReduceAll::init() {
	_outputs[0]->initValue({ 1, 1, 1, 1});
	_reduce_op = _param->reduce_all_param().reduce_op();
	std::string op_name = (_reduce_op == deepflow::ReduceAllParam_ReduceAllOp_AVG ? "reduce_mean" : "reduce_sum");
	_outputs[0]->initDiff();
	LOG(INFO) << op_name << " " << _name << " - " << _outputs[0]->value()->shape();
}

void ReduceAll::forward() {
	auto size = _inputs[0]->value()->size();
	DF_CUDA_CHECK(hipMemset(_outputs[0]->value()->mutableData(), 0, _outputs[0]->value()->sizeInBytes()));	
	ReduceAllKernelForward << < numOfBlocks(size), maxThreadsPerBlock >> > (size, _reduce_op == deepflow::ReduceAllParam_ReduceAllOp_AVG, (float*)_inputs[0]->value()->data(), (float*)_outputs[0]->value()->mutableData());
	DF_KERNEL_CHECK();
}

void ReduceAll::backward() {
	auto size = _inputs[0]->value()->size();
	ReduceAllKernelBackward << < numOfBlocks(size), maxThreadsPerBlock >> > (size, _reduce_op == deepflow::ReduceAllParam_ReduceAllOp_AVG, (float*)_outputs[0]->diff()->data(), (float*)_inputs[0]->diff()->mutableData());
	DF_KERNEL_CHECK();
}

std::string ReduceAll::to_cpp() const
{
	return "";
}