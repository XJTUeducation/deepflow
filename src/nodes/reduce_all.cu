#include "hip/hip_runtime.h"
#include "nodes/reduce_all.h"
#include "core/common_cu.h"

__global__
void ReduceAllKernelForward(const int n, bool average, const float *x, float *y)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i < n) {		
		atomicAdd(&y[0], average ? x[i] / n : x[i]);
	}
}

__global__
void ReduceAllKernelBackward(const int n, bool average, const float *dY, float *dX)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i < n) {
		dX[i] = (average ? dY[0] / n : dY[0]);
	}		
}

ReduceAll::ReduceAll(deepflow::NodeParam *param) : Node(param) {
	LOG_IF(FATAL, param->has_reduce_all_param() == false) << "param.has_reduce_all_param() == false";
	_reduce_op = _param->reduce_all_param().reduce_op();
}

std::string ReduceAll::op_name() const
{	
	std::string op_name = (_reduce_op == deepflow::ReduceAllParam_ReduceAllOp_AVG ? "reduce_mean" : "reduce_sum");
	return op_name;
}

void ReduceAll::init() {
	_outputs[0]->initValue({ 1, 1, 1, 1});	
	_outputs[0]->initDiff();	
}

void ReduceAll::forward() {
	auto size = _inputs[0]->value()->size();
	DF_CUDA_CHECK(hipMemset(_outputs[0]->value()->gpu_data(DF_LINE), 0, _outputs[0]->value()->bytes()));	
	ReduceAllKernelForward << < numOfBlocks(size), maxThreadsPerBlock >> > (size, _reduce_op == deepflow::ReduceAllParam_ReduceAllOp_AVG, _inputs[0]->value()->gpu_data(DF_LINE), (float*)_outputs[0]->value()->gpu_data(DF_LINE));
	DF_KERNEL_CHECK();
}

void ReduceAll::backward() {
	if (_inputs[0]->diff()) {
		auto size = _inputs[0]->value()->size();
		ReduceAllKernelBackward << < numOfBlocks(size), maxThreadsPerBlock >> > (size, _reduce_op == deepflow::ReduceAllParam_ReduceAllOp_AVG, _outputs[0]->diff()->gpu_data(DF_LINE), (float*)_inputs[0]->diff()->gpu_data(DF_LINE));
		DF_KERNEL_CHECK();
	}
}

std::string ReduceAll::to_cpp() const
{
	return "";
}