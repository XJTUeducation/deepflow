#include "hip/hip_runtime.h"
#include "solvers/adadelta_solver.h"
#include "core/common_cu.h"
#include "nodes/variable.h"

#include <glog/logging.h>

__global__
void FillKernel(int n, float *a)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i < n) a[i] = 1.0f;
}

__global__
void AdaDeltaKernel(const int n, float *w, const float *g, float *h1, float *h2, const float momentum, const float learning_rate, const float delta)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i < n) {
		float gi = g[i];
		float hi = h1[i] = momentum * h1[i] + (1 - momentum) * gi * gi;
		gi = gi * sqrt((h2[i] + delta) / (hi + delta));
		h2[i] = momentum * h2[i] + (1 - momentum) * gi * gi;
		w[i] -= learning_rate * gi;
	}
}

AdaDeltaSolver::AdaDeltaSolver(deepflow::SolverParam *param) : Solver(param) {
	LOG_IF(FATAL, param->has_adadelta_solver() == false) << "param.has_adadelta_solver() == false";
	_my_param = param->mutable_adadelta_solver();
	_learning_rate = param->learning_rate();
}

void AdaDeltaSolver::apply(std::shared_ptr<Variable> var, hipStream_t stream) {
	auto context = var->executionContext();
	bool verbos = (context && context->debug_level > 3) ? true : false;
	if (_initialized == false) {
		LOG_IF(INFO, verbos) << "solver " << name() << " for variable " << var->name();
		init(var);
	}
	if (!_enabled)
		return;
	LOG_IF(INFO, verbos) << "applying solver " << name() << " ON " << var->name();	
	auto size = var->output(0)->value()->size();
	AdaDeltaKernel << <numOfBlocks(size), maxThreadsPerBlock, 0, stream >> > (size, (float*)var->output(0)->value()->mutableData(), (float*)var->gradients(), _h1, _h2, _my_param->momentum(), _learning_rate, _my_param->delta());
	DF_KERNEL_CHECK();
	var->reset_gradients(stream);
}

void AdaDeltaSolver::init(std::shared_ptr<Variable> var) {
	auto size = var->output(0)->value()->size();
	auto sizeInBytes = var->output(0)->value()->sizeInBytes();
	DF_CUDA_CHECK(hipMalloc(&_h1, sizeInBytes));	
	FillKernel << <numOfBlocks(size), maxThreadsPerBlock >> >(size, _h1);
	DF_KERNEL_CHECK();
	DF_CUDA_CHECK(hipMalloc(&_h2, sizeInBytes));	
	FillKernel << <numOfBlocks(size), maxThreadsPerBlock >> >(size, _h2);
	DF_KERNEL_CHECK();
	_initialized = true;
}

std::string AdaDeltaSolver::to_cpp() const
{	
	std::string cpp = "auto " + name() + " = df.adadelta_solver(";
	cpp += std::to_string(_param->learning_rate()) + ", ";
	cpp += std::to_string(_my_param->momentum()) + ", ";
	cpp += std::to_string(_my_param->delta()) + ", ";
	cpp += "\"" + name() + "\"";
	cpp += ");";
	return cpp;
}
