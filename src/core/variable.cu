#include "hip/hip_runtime.h"

#include "core/variable.h"
#include "core/initializer.h"

#include <string>
#include <iostream>

#include "core/common_cu.h"

#include <opencv2/opencv.hpp>

#include <glog/logging.h>

Variable::Variable(std::shared_ptr<Initializer> initializer, const NodeParam &param) : Node(param) {
	LOG_IF(FATAL, param.has_variable_param() == false) << "param.has_variable_param() == false";
	_initializer = initializer;			
}

void Variable::initForward() {	
	_initializer->init();
	_outputs[0]->initValue(_initializer->dims());
	LOG(INFO) << "Initializing Variable " << _name << " - " << _outputs[0]->value()->shape();
	if (_param.variable_param().has_weights()) {
		LOG_IF(FATAL, hipMemcpy(
			_outputs[0]->value()->mutableData(),
			_param.variable_param().weights().weight().data(),
			_outputs[0]->value()->sizeInBytes()
			, hipMemcpyHostToDevice) != 0) << "hipMemcpy [FAILED]";
	}
	else if (_initializer->param().has_init_data())
		LOG_IF(FATAL,hipMemcpy(
			_outputs[0]->value()->mutableData(),
			_initializer->param().init_data().weight().data(),
			_outputs[0]->value()->sizeInBytes()
			,hipMemcpyHostToDevice) != 0) << "hipMemcpy [FAILED]";		
	else {
		_initializer->apply(this);
		for (int i = 0; i < _outputs[0]->value()->size(); ++i)
			_param.mutable_variable_param()->mutable_init_param()->mutable_init_data()->add_weight(0);
		LOG_IF(FATAL, hipMemcpy(			
			_param.mutable_variable_param()->mutable_init_param()->mutable_init_data()->mutable_weight()->mutable_data(),
			_outputs[0]->value()->data(),
			_outputs[0]->value()->sizeInBytes()
			, hipMemcpyDeviceToHost) != 0) << "hipMemcpy [FAILED]";
	}
}

void Variable::forward() {
	
}

void Variable::initBackward() {	
	_outputs[0]->initDiff();
}

bool Variable::snapshot() {
	return _param.variable_param().has_snapshot_param();
}

int Variable::snapshotInterval() {
	return _param.variable_param().snapshot_param().snapshot_interval();
}

void Variable::transferDataToParam() {
	auto dma = _param.mutable_variable_param()->mutable_weights();
	for (int i = 0; i < _outputs[0]->value()->size(); ++i)
		dma->add_weight(0);
	LOG_IF(FATAL,
		hipMemcpy(
			dma->mutable_weight()->mutable_data(),
			_outputs[0]->value()->data(),
			_outputs[0]->value()->sizeInBytes(), hipMemcpyDeviceToHost
		) != 0) << "hipMemcpy [FAILED]";
}

void Variable::toImage(int iteration) {	
	LOG_IF(FATAL, _param.variable_param().has_snapshot_param() == false);
	const SnapshotParam &param = _param.variable_param().snapshot_param();
	int perImageWidth = param.per_image_width();
	int perImageHeight = param.per_image_height();
	std::string imagePath = param.snapshot_prefix() + std::to_string(iteration) + ".png";
	LOG(INFO) << _name << " to image " << imagePath;	
	auto weights = _outputs[0]->value()->cpyToHost<float>();
	auto dims = _outputs[0]->value()->dims();
	int numImages = dims[1];
	int perImage = dims[0];
	int sq = (int)floor(sqrt((float)numImages));
	int picWidth = perImageWidth * sq;
	int picHeight = perImageHeight * ((int)ceil(((float)numImages / sq)));
	for (int c = 0; c < numImages; ++c)
	{
		float max = -FLT_MAX;
		float min =  FLT_MAX;
		for (int r = 0; r < perImage; ++r)
		{
			float tmp = weights->at(r*numImages + c);
			if (max < tmp)
				max = tmp;
			if (min > tmp)
				min = tmp;
		}
		for (int r = 0; r < perImage; ++r)
		{
			float &tmp = weights->at(r*numImages + c);
			tmp = (tmp - min) / (max-min) * 255;
		}
	}
	cv::Mat result(picHeight, picWidth, CV_32SC1);
	result = 0;
	for (int c = 0; c < numImages; ++c)
	{
		for (int r = 0; r < perImage; ++r)
		{
			float &tmp = weights->at(r*numImages + c);
			int csmall = r % perImageWidth;
			int rsmall = (r - csmall) / perImageWidth;
			int cbig = c % sq;
			int rbig = (c - cbig) / sq;
			int nc = cbig*perImageWidth + csmall;
			int nr = rbig*perImageHeight + rsmall;
			result.at<int>(nr, nc) = tmp;
		}
	}
	cv::imwrite(imagePath, result);
}

/*
__global__
void PlaceHolderImageKernel(const int n,const float *in, int *out, const int picWidth, const int sq, const int numImages, const int perImageWidth, const int perImageHeight)
{
int i = blockIdx.x*blockDim.x + threadIdx.x;
if (i < n) {
int flat_pixel = i % (perImageWidth * perImageHeight);
int num_image = (i-flat_pixel) / (perImageWidth * perImageHeight);
int csmall = flat_pixel % perImageWidth;
int rsmall = (flat_pixel - csmall) / perImageWidth;
int cbig = num_image % sq;
int rbig = (num_image - cbig) / sq;
int c = cbig*perImageWidth + csmall;
int r = rbig*perImageHeight + rsmall;
out[r*picWidth+c] = (in[i] + 1.0f) / 2.0f * 255;
}
}

void PlaceHolder::toImage(std::string imagePath) {
LOG(INFO) << _name << " to image " << imagePath;
auto shape = _outputs[0]->shape();
int numImages = shape->at(1);
int perImageHeight = shape->at(2);
int perImageWidth = shape->at(3);
int sq = (int)floor(sqrt((float)numImages));
int picWidth = perImageWidth * sq;
int picHeight = perImageHeight * ((int)ceil(((float)numImages / sq)));
int n = picWidth * picHeight;
int *d_img;
LOG_IF(FATAL, hipMalloc(&d_img, sizeof(int) * n) != 0);
LOG_IF(FATAL, hipMemset(d_img, 0, sizeof(int) * n) != 0);
PlaceHolderImageKernel << < numOfBlocks(shape->size()), maxThreadsPerBlock >> >(shape->size(), _outputs[0]->value(), d_img, picWidth, sq, numImages,perImageWidth, perImageHeight);
LOG_IF(FATAL, hipPeekAtLastError() != 0);
int *img = new int[n];
LOG_IF(FATAL, hipMemcpy(img, d_img, sizeof(int) *n,hipMemcpyDeviceToHost) != 0);
LOG_IF(FATAL, hipFree(d_img) != 0);
cv::Mat imgMat(picHeight,picWidth,CV_32SC1,img);
cv::imwrite(imagePath, imgMat);
delete[] img;
}

*/