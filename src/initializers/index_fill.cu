#include "hip/hip_runtime.h"
#include "core/common_cu.h"

#include "initializers/index_fill.h"
#include "core/variable.h"

__global__
void IndexFillKernel(const int n, float *a,const float offset)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i < n) a[i] = offset + i;
}

IndexFill::IndexFill(const InitParam &param) : Initializer(param) {
	LOG_IF(FATAL, param.has_index_fill_param() == false);	
}

void IndexFill::apply(Variable *variable) {
	float offset = _param.index_fill_param().offset();
	auto size = variable->output(0)->value()->size();
	LOG(INFO) << "Filling variable " << variable->name() << " with " << offset;
	IndexFillKernel << <numOfBlocks(size), maxThreadsPerBlock >> >(size, (float*)variable->output(0)->value()->mutableData(), offset);
	LOG_IF(FATAL, hipPeekAtLastError() != 0);
}