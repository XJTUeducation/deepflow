#include "hip/hip_runtime.h"
#include "core/common_cu.h"

#include "ops/bias_add.h"

__global__
void BiasAddKernelForward(const int n, const float *a, const int bias_dim, const float *b, float *c)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i < n) c[i] = a[i] + b[i%bias_dim];
}

__global__
void BiasAddKernelBackward(const int n, const float *diff, const int num_samples, const int bias_dim, float *bias_diff)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i < n)
	{
		int j = i%bias_dim;		
		atomicAdd(&bias_diff[j], diff[i] / num_samples);
		//bias_diff[j] = bias_diff[j] + diff[i] / num_samples;
	}
}

BiasAdd::BiasAdd(const NodeParam &param) : Node(param) {
	LOG_IF(FATAL, param.has_op_bias_add_param() == false);
}

void BiasAdd::initForward() {
	auto a = _inputs[0];
	auto ad = a->dims();

	auto b = _inputs[1];
	auto bd = b->dims();

	LOG_IF(FATAL, ad[1] != bd[1]) << "ad[1] != bd[1] [FAILED]";
	_outputs[0]->initValue({ ad[0], ad[1], 1, 1 });	
	LOG(INFO) << "Initializing Bias (name: " << _name << " ) - " << a->value()->toString() << " + " << b->value()->toString() << " -> " << _outputs[0]->value()->toString();	
}

void BiasAdd::initBackward() {
	_outputs[0]->initDiff();
}

void BiasAdd::forward() {
	// C(m,n) = A(m,n) + B(m,n)	
	auto outputDims = _outputs[0]->value()->dims();	
	auto size = _outputs[0]->value()->size();
	auto bias_dim = outputDims[1];
	BiasAddKernelForward <<< numOfBlocks(size), maxThreadsPerBlock >>> (size, (float*) _inputs[0]->value()->data(), bias_dim, (float*) _inputs[1]->value()->data(), (float*) _outputs[0]->value()->mutableData());
	LOG_IF(FATAL, hipPeekAtLastError() != 0);	
}

void BiasAdd::backward() {
	LOG_IF(FATAL, hipMemcpy( _inputs[0]->diff()->mutableData(), _outputs[0]->diff()->data(), _inputs[0]->diff()->sizeInBytes(), hipMemcpyDeviceToDevice) != 0);
	LOG_IF(FATAL, hipMemset( _inputs[1]->diff()->mutableData(), 0, _inputs[1]->diff()->sizeInBytes()) != 0);
	auto outputDims = _outputs[0]->diff()->dims();	
	auto size = _outputs[0]->diff()->size();
	BiasAddKernelBackward <<< numOfBlocks(size), maxThreadsPerBlock >>> (size, (float*)_outputs[0]->diff()->data(), outputDims[0], outputDims[1], (float*)_inputs[1]->diff()->mutableData());
	LOG_IF(FATAL, hipPeekAtLastError() != 0);
}
